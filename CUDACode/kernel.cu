#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <iostream>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <cmath>
#include <chrono>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
using std::chrono::duration;
using std::chrono::duration_cast;
using std::chrono::high_resolution_clock;
using std::milli;

using namespace std;

//____________________ NOTES__________________________________
// ___________________________________________________________
// for our pc cache size is 6 MB
// GPU is 256 MB
// => dataset sizes are (cause we use hipfftComplex(8) arrays):
// + cache:  max 1572864 / 4 for each vector
// + GPU:  max 67108864 / 4 for each vector

int N = 67108864 / 4;

// Data padding for fitting in after calculations
void PadData(const hipfftComplex*, hipfftComplex**, int);

// Normalization for number after multiplipaction
void Normalize(hipfftComplex*, hipfftComplex**, int);

// Complex multiplication
static __device__ __host__ inline hipfftComplex ComplexMul(hipfftComplex, hipfftComplex);
// Complex pointwise multiplication
static __global__ void ComplexPointwiseMul(hipfftComplex*, const hipfftComplex*, int);


int main(int argc, char* argv[]) {

	hipfftComplex* vec1_padded, * vec1, * vec2, * vec2_padded;
	vec1 = (hipfftComplex*)malloc(N * sizeof(hipfftComplex));
	vec2 = (hipfftComplex*)malloc(N * sizeof(hipfftComplex));

	// initialization
	for (int i = 0; i < N; i++)
	{
		vec1[i].x = 1;
		vec1[i].y = 0;
		vec2[i].x = 1;
		vec2[i].y = 0;
	}

	auto startTime = high_resolution_clock::now();
	// padding data
	size_t new_size = 1;
	while (new_size < N)
		new_size <<= 1;
	new_size <<= 1;
	PadData(vec1, &vec1_padded, new_size);
	PadData(vec2, &vec2_padded, new_size);

	hipFree(vec1);
	hipFree(vec2);

	// save arrays from host to device
	hipfftComplex* vec1_DEVICE;
	hipMalloc((void**)&vec1_DEVICE, new_size * sizeof(hipfftComplex));
	hipMemcpy(vec1_DEVICE, vec1_padded, new_size * sizeof(hipfftComplex), hipMemcpyHostToDevice);

	hipfftComplex* vec2_DEVICE;
	hipMalloc((void**)&vec2_DEVICE, new_size * sizeof(hipfftComplex));
	hipMemcpy(vec2_DEVICE, vec2_padded, new_size * sizeof(hipfftComplex), hipMemcpyHostToDevice);

	hipFree(vec1_padded);
	hipFree(vec2_padded);

	hipfftHandle plan_VEC1;
	hipfftPlan1d(&plan_VEC1, new_size, HIPFFT_C2C, 1);
	hipfftExecC2C(plan_VEC1, vec1_DEVICE, vec1_DEVICE, HIPFFT_FORWARD);

	hipfftHandle plan_VEC2;
	hipfftPlan1d(&plan_VEC2, new_size, HIPFFT_C2C, 1);
	hipfftExecC2C(plan_VEC2, vec2_DEVICE, vec2_DEVICE, HIPFFT_FORWARD);

	// Multiplication
	ComplexPointwiseMul << <32, 256 >> > (vec1_DEVICE, vec2_DEVICE, new_size * sizeof(hipfftComplex));
	hipfftExecC2C(plan_VEC1, vec1_DEVICE, vec1_DEVICE, HIPFFT_BACKWARD);

	hipfftComplex* test_out;
	test_out = (hipfftComplex*)malloc(new_size * sizeof(hipfftComplex));
	hipMemcpy(test_out, vec1_DEVICE, new_size * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
	hipfftComplex* test_out2;
	Normalize(test_out, &test_out2, new_size);


	/*for (int i = 0; i < new_size; i++)
	{
		cout << test_out2[i].x;
		cout << endl;
	}
	cout << endl << "____________" << endl;*/


	auto endTime = high_resolution_clock::now();
	auto overallTime = duration_cast<duration<double, milli>>(endTime - startTime).count();
	cout << overallTime << " ms" << '\n';

	hipFree(vec1_DEVICE);
	hipFree(vec2_DEVICE);
	hipFree(test_out);
	hipFree(test_out2);
	hipfftDestroy(plan_VEC1);
	hipfftDestroy(plan_VEC2);
}

// Complex multiplication
static __device__ __host__ inline hipfftComplex ComplexMul(hipfftComplex a, hipfftComplex b)
{
	hipfftComplex c;
	c.x = a.x * b.x - a.y * b.y;
	c.y = a.x * b.y + a.y * b.x;
	return c;
}


// Complex pointwise multiplication
static __global__ void ComplexPointwiseMul(hipfftComplex* a, const hipfftComplex* b, int size)
{
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	for (int i = threadID; i < size; i += numThreads)
		a[i] = ComplexMul(a[i], b[i]);
}


void PadData(const hipfftComplex* signal, hipfftComplex** padded_signal, int new_size) {
	hipfftComplex* new_data = reinterpret_cast<hipfftComplex*>(malloc(sizeof(hipfftComplex) * new_size));
	memcpy(new_data + 0, signal, N * sizeof(hipfftComplex));
	memset(new_data + N, 0, (new_size - N) * sizeof(hipfftComplex));
	*padded_signal = new_data;
}

void Normalize(hipfftComplex* signal, hipfftComplex** normalized_signal, int new_size) {
	hipfftComplex* new_data = reinterpret_cast<hipfftComplex*>(malloc(sizeof(hipfftComplex) * new_size));
	int zeros = 0;
	for (int i = new_size - 1; i >= 0; i--)
		if (round(signal[i].x) == 0) zeros++;
		else break;
	memcpy(new_data + zeros, signal, (new_size - zeros) * sizeof(hipfftComplex));
	memset(new_data + 0, 0, zeros * sizeof(hipfftComplex));
	long long carry = 0;
	for (size_t i = 0; i < new_size; ++i) {
		new_data[i].x = round(new_data[i].x / new_size);
		new_data[i].x += carry;
		carry = new_data[i].x / 10;
		new_data[i].x = (int)(new_data[i].x) % 10;
	}
	*normalized_signal = new_data;
}